#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float maxval = *input_a;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_a[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_a[x] - maxval);
  }
  // Compute per-row loss.
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}

__global__ void matrix_softmax_kernel(int nrow, int ncol, const float *input_a, float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  output += y * ncol;
  float maxval = *input_a;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_a[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_a[x] - maxval);
  }
  for (int x = 0; x < ncol; ++x) {
    output[x] = exp(input_a[x] - maxval) / sum;
  }
}

__global__ void array_relu_kernel(int64_t length, const float* input1, float *output) {
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if(y >= length) {
    return;
  }
  output[y] = max(0.0f, input1[y]);
}

__global__ void array_relu_gradient_kernel(int64_t length, const float* input1, const float* in_grad, float *output) {
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if(y >= length) {
    return;
  }
  output[y] = input1[y] >= 0.0f ? in_grad[y] : 0.0f;
}

__global__ void array_add_kernel(int64_t length, const float* input1, const float* input2, float *output) {
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if(y >= length) {
    return;
  }
  output[y] = input1[y] + input2[y];
}

__global__ void array_add_by_const_kernel(int64_t length, const float* input1, float value, float *output) {
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if(y >= length) {
    return;
  }
  output[y] = input1[y] + value;
}

__global__ void array_mul_kernel(int64_t length, const float* input1, const float* input2, float *output) {
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if(y >= length) {
    return;
  }
  output[y] = input1[y] * input2[y];
}

__global__ void array_mul_by_const_kernel(int64_t length, const float* input1, float value, float *output) {
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if(y >= length) {
    return;
  }
  output[y] = input1[y] * value;
}

__global__ void array_set_kernel(int64_t length, float value, float *output) {
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if(y >= length) {
    return;
  }
  output[y] = value;
}

int DLGpuArraySet(DLArrayHandle arr, float value) { /* TODO: Your code here */
  dim3 threads;
  dim3 blocks;
  int64_t length = 1;
  for(int i = 0; i < arr->ndim; i++) {
    length *= arr->shape[i];
  }
  if(length <= 1024) {
    array_set_kernel<<<1, length>>>(length, value, (float*)arr->data);
  } else {
    array_set_kernel<<<length / 1024 + 1, 1024>>>(length, value, (float*)arr->data);
  }
  return 0;
}

__global__ void broadcast_to_kernel(int64_t input_length, const float* input, float *output) {
  output += input_length * blockIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if(x < input_length) {
    output[x] = input[x];
  }
}

int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  dim3 threads;
  dim3 blocks;
  int64_t input_length = 1;
  for(int i = 0; i < input->ndim; i++) {
    input_length *= input->shape[i];
  }
  broadcast_to_kernel<<<dim3(input_length / 1024 + 1, output->shape[0]), 1024>>>(input_length, (const float*)input->data, (float*)output->data);
  return 0;
}


__global__ void reduce_sum_axis_zero_kernel(int64_t output_length, int reduce_size, const float* input, float *output) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  if(x >= output_length) return;
  float value = 0;
  for(int i = 0; i < reduce_size; i++) {
    value += input[i * output_length + x];
    __syncthreads();
  }
  output[x] = value;
}


int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  int output_length = 1;
  for(int i = 0; i < output->ndim; i++) {
    output_length *= output->shape[i];
  }
  printf("DLGpuReduceSumAxisZero : %d, %d\n", output_length, input->shape[0]);
  reduce_sum_axis_zero_kernel<<<output_length / 1024 + 1, min(1024, output_length)>>>(output_length, input->shape[0], (float*)input->data, (float*)output->data);
  return 0;
}

int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, DLArrayHandle output) {
  /* TODO: Your code here */
  dim3 threads;
  dim3 blocks;
  int64_t length = 1;
  for(int i = 0; i < output->ndim; i++) {
    length *= output->shape[i];
  }
  if(length <= 1024) {
    array_add_kernel<<<1, length>>>(length, (const float*)matA->data, (const float*)matB->data, (float*)output->data);
  } else {
    array_add_kernel<<<length / 1024 + 1, 1024>>>(length, (const float*)matA->data, (const float*)matB->data, (float*)output->data);
  }
  return 0;
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
  /* TODO: Your code here */
  dim3 threads;
  dim3 blocks;
  int64_t length = 1;
  for(int i = 0; i < output->ndim; i++) {
    length *= output->shape[i];
  }
  if(length <= 1024) {
    array_add_by_const_kernel<<<1, length>>>(length, (const float*)input->data, val, (float*)output->data);
  } else {
    array_add_by_const_kernel<<<length / 1024 + 1, 1024>>>(length, (const float*)input->data, val, (float*)output->data);
  }
  return 0;
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output) {
  /* TODO: Your code here */
  dim3 threads;
  dim3 blocks;
  int64_t length = 1;
  for(int i = 0; i < output->ndim; i++) {
    length *= output->shape[i];
  }
  if(length <= 1024) {
    array_mul_kernel<<<1, length>>>(length, (const float*)matA->data, (const float*)matB->data, (float*)output->data);
  } else {
    array_mul_kernel<<<length / 1024 + 1, 1024>>>(length, (const float*)matA->data, (const float*)matB->data, (float*)output->data);
  }
  return 0;
}

int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                               DLArrayHandle output) {
  /* TODO: Your code here */
  dim3 threads;
  dim3 blocks;
  int64_t length = 1;
  for(int i = 0; i < output->ndim; i++) {
    length *= output->shape[i];
  }
  if(length <= 1024) {
    array_mul_by_const_kernel<<<1, length>>>(length, (const float*)input->data, val, (float*)output->data);
  } else {
    array_mul_by_const_kernel<<<length / 1024 + 1, 1024>>>(length, (const float*)input->data, val, (float*)output->data);
  }
  return 0;
}

int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
  /* TODO: Your code here */
  // Hint: use cublas
  // cublas assume matrix is column major
  // op(A) * op(B) = C
  // op(B)T * op(A)T = CT

  hipblasHandle_t handle;
  float one = 1.0f;
  float zero = 0.0f;
  int m = matC->shape[1];
  int n = matC->shape[0];
  int k = transposeA ? matA->shape[0] : matA->shape[1];
  hipblasCreate(&handle);
  hipblasSgemm(handle,
    transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
    transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
    m, n, k,
    &one,
    (const float*)matB->data, !transposeB ? m : k,
    (const float*)matA->data, !transposeA ? k : n,
    &zero,
    (float*)matC->data, m
  );
  return 0;
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  dim3 threads;
  dim3 blocks;
  int64_t length = 1;
  for(int i = 0; i < output->ndim; i++) {
    length *= output->shape[i];
  }
  if(length <= 1024) {
    array_relu_kernel<<<1, length>>>(length, (const float*)input->data, (float*)output->data);
  } else {
    array_relu_kernel<<<length / 1024 + 1, 1024>>>(length, (const float*)input->data, (float*)output->data);
  }
  return 0;
}

int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                      DLArrayHandle output) {
  /* TODO: Your code here */
  dim3 threads;
  dim3 blocks;
  int64_t length = 1;
  for(int i = 0; i < output->ndim; i++) {
    length *= output->shape[i];
  }
  if(length <= 1024) {
    array_relu_gradient_kernel<<<1, length>>>(length, (const float*)input->data, (const float*)in_grad->data, (float*)output->data);
  } else {
    array_relu_gradient_kernel<<<length / 1024 + 1, 1024>>>(length, (const float*)input->data, (const float*)in_grad->data, (float*)output->data);
  }
  return 0;
}

int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  assert(input->ndim == 2);
  assert(output->ndim == 2);
  int nrow = input->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = input->shape[1];
  const float *input_data_a = (const float *)input->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_softmax_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, input_data_a, output_data);
  return 0;
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
                             const DLArrayHandle input_b,
                             DLArrayHandle output) {
  assert(input_a->ndim == 2);
  assert(input_b->ndim == 2);
  assert(output->ndim == 1);
  assert(input_a->shape[0] == input_b->shape[0] &&
         input_a->shape[1] == input_b->shape[1]);
  int nrow = input_a->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = input_a->shape[1];
  const float *input_data_a = (const float *)input_a->data;
  const float *input_data_b = (const float *)input_b->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_softmax_cross_entropy_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, input_data_a, input_data_b, output_data);
  return 0;
}
